#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <cassert>
#include <hip/hip_runtime.h>
#include <chrono>
#include <unistd.h>
#include <iostream>
#include <mpi.h>

#define DEV_NO 0
#define ROUND_MAX 4

const int INF = 1000000000;
const int V = 20010;
void input(char *inFileName);
void output(char *outFileName);

void block_FW(int B);
int ceil(int a, int b);
bool cal(int B, int Round, int block_start_x, int block_start_y, int block_width, int block_height);
__global__ void cal_kernel(int B, int Round, int block_start_x, int block_start_y, int block_width, int block_height, int n, int* Dist_gpu);
__global__ void p1_cal_kernel(int B, int Round, int block_start_x, int block_start_y, int block_width, int block_height, int n, int* Dist_gpu, int pitch_int);
__global__ void p2_cal_kernel(int B, int Round, int n, int* Dist_gpu, int pitch_int);
/*MPI*/
__global__ void p3_cal_kernel(int B, int Round, int k_i, int n, int* Dist_gpu, int pitch_int);


int n, m;	// Number of vertices, edges
int* Dist;
int* Dist_gpu;
hipDeviceProp_t prop;
size_t pitch;

/*MPI*/
int num_gpus;
int rank, size;
int chunk, more;
int proc_start_b_i, proc_end_b_i;
int valid_row;


int main(int argc, char* argv[])
{	
	assert(argc==4);
	hipGetDeviceProperties(&prop, DEV_NO);
    hipGetDeviceCount(&num_gpus);

	input(argv[1]);
	int B = atoi(argv[3]);
	assert((B*B-1)/prop.maxThreadsPerBlock < ROUND_MAX);
    /*MPI*/
    int Round = ceil(n, B);

    /*MPI*/
    MPI_Init(&argc, &argv);
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &size);
    chunk = Round / size;
    more = Round % size;
    proc_start_b_i = (rank<more)? (chunk+1)*rank : (chunk+1)*more + chunk*(rank-more);
    proc_end_b_i = (rank<more)? proc_start_b_i + (chunk+1) : proc_start_b_i + chunk;
    valid_row = (proc_end_b_i*B > n)? n - proc_start_b_i*B : (proc_end_b_i - proc_start_b_i)*B;
    printf("%d: %d %d\n", rank, proc_start_b_i, proc_end_b_i);
    
	//auto start = std::chrono::high_resolution_clock::now();
	block_FW(B);
	//auto end = std::chrono::high_resolution_clock::now();
	//std::chrono::duration<double> diff = end - start;
    //std::cout << rank << ": " << diff.count() * 1000 << "(ms)\n";
    //MPI_Barrier(MPI_COMM_WORLD);
    
    int rc;
    MPI_File fh;
    MPI_Status st;
    rc = MPI_File_open(MPI_COMM_WORLD, argv[2], MPI_MODE_CREATE | MPI_MODE_WRONLY, MPI_INFO_NULL, &fh);
    //MPI_Barrier(MPI_COMM_WORLD);
    //MPI_File_write_at(fh, sizeof(int)*proc_start_b_i*B*n, Dist*proc_start_b_i*B*n, valid_row*n, MPI_INT, &st);
    MPI_File_write_at(fh, sizeof(int)*proc_start_b_i*B*n, Dist+proc_start_b_i*B*n, valid_row*n, MPI_INT, &st);
    MPI_File_close(&fh);
    //MPI_Finalize();
	//output(argv[2]);

	return 0;
}

void input(char *inFileName)
{
    FILE *infile = fopen(inFileName, "r");
    fseek(infile, 0, SEEK_END);
    long lsize = ftell(infile);
    rewind(infile);

    char* input_buff = (char*) malloc(sizeof(char)*lsize);
    assert(input_buff != NULL);
    size_t result = fread(input_buff, 1, lsize, infile);
    assert(result == lsize);
    n = atoi(strtok(input_buff, " \n"));
    m = atoi(strtok(NULL, " \n"));


    //Dist = (int*)malloc(n*n*sizeof(int));
    hipHostMalloc(&Dist, sizeof(int)*n*n);
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            if (i == j) Dist[i*n + j] = 0;
            else        Dist[i*n + j] = INF;
        }
    }
    //printf("%d %d\n", n, m);

    while (--m >= 0) {
        int a, b, v;
        a = atoi(strtok(NULL, " \n"));
        b = atoi(strtok(NULL, " \n"));
        v = atoi(strtok(NULL, " \n"));
        Dist[a*n + b] = v;
    }
    fclose(infile);
    free(input_buff);
}

void output(char *outFileName)
{
	FILE *outfile = fopen(outFileName, "w");
	fwrite(Dist, sizeof(int), n*n, outfile);
    fclose(outfile);
}

int ceil(int a, int b)
{
	return (a + b -1)/b;
}

void block_FW(int B)
{
    /*MPI*/
    unsigned int g_id = rank % num_gpus;
    hipSetDevice(g_id); 

	int num_thread = (B*B>prop.maxThreadsPerBlock)? prop.maxThreadsPerBlock: B*B; 
	int round = ceil(n, B);
	hipMallocPitch((void**)&Dist_gpu, &pitch,n*sizeof(int), n+128);    /*MPI*/
	int pitch_int = pitch / sizeof(int);
	hipMemcpy2D(Dist_gpu, pitch, Dist, n*sizeof(int), n*sizeof(int), n, hipMemcpyHostToDevice);

    /*MPI*/
    //MPI_Status st;
    MPI_Request req;
    int sr_buff[n*B];
    dim3 grid3_p(proc_end_b_i-proc_start_b_i, round-1);	

	dim3 grid2(round-1, 2);
	dim3 grid3(round-1, round-1);
	dim3 block(B, num_thread/B);
	for (int r = 0; r < round; ++r) {
        //printf("%d %d\n", r, round);
		/* Phase 1*/
		p1_cal_kernel<<< 1, block, B*B*sizeof(int) >>>(B, r,	r,	r,	1,	1, n, Dist_gpu, pitch_int);

		/* Phase 2*/
		p2_cal_kernel<<< grid2, block, 2*B*B*sizeof(int) >>>(B, r, n, Dist_gpu, pitch_int); 
		// cal_kernel<<<           r*1, num_thread >>>(B, r,     r,     0,             r,             1, n, Dist_gpu);
		// cal_kernel<<< (round-r-1)*1, num_thread >>>(B, r,     r,  r +1,  round - r -1,             1, n, Dist_gpu);
		// cal_kernel<<<           1*r, num_thread >>>(B, r,     0,     r,			 1, 			r, n, Dist_gpu);
		// cal_kernel<<< 1*(round-r-1), num_thread >>>(B, r,  r +1,     r,             1,  round - r -1, n, Dist_gpu);

		/* Phase 3*/
			
		//p3_cal_kernel<<< grid3_p, block>>>(B, r, proc_start_b_i, n, Dist_gpu, pitch_int);
		p3_cal_kernel<<< grid3, block>>>(B, r, 0, n, Dist_gpu, pitch_int);
        
        /*MPI*/
        //int more = round%size;
        /*if(r+1 < round) {
        int sender_rank = (r+1 < more*(chunk+1))? (r+1)/(chunk+1) :  more + (r+1-more*(chunk+1))/chunk;
        //printf("%d %d %d rank:%d sender_rank:%d\n", more, chunk, r, rank, sender_rank);
        if(rank == sender_rank) hipMemcpy2D(sr_buff, n*sizeof(int), Dist_gpu+(r+1)*B*pitch_int, pitch, n*sizeof(int), B, hipMemcpyDeviceToHost); 
        MPI_Bcast(sr_buff, n*B, MPI_INT, sender_rank, MPI_COMM_WORLD);
        if(rank != sender_rank) hipMemcpy2D(Dist_gpu+(r+1)*B*pitch_int, pitch, sr_buff, n*sizeof(int), n*sizeof(int), B, hipMemcpyHostToDevice);
        }*/

		// cal_kernel<<<                     r*r, num_thread >>>(B, r,     0,     0,            r,             r, n, Dist_gpu);
		// cal_kernel<<<           (round-r-1)*r, num_thread >>>(B, r,     0,  r +1,  round -r -1,             r, n, Dist_gpu);
		// cal_kernel<<<           r*(round-r-1), num_thread >>>(B, r,  r +1,     0,            r,  round - r -1, n, Dist_gpu);
		// cal_kernel<<< (round-r-1)*(round-r-1), num_thread >>>(B, r,  r +1,  r +1,  round -r -1,  round - r -1, n, Dist_gpu);
		
	}
    
    /*MPI*/
    //printf("%d, %d", Dist, Dist_gpu);
	hipMemcpy2D(Dist+proc_start_b_i*B*n, n*sizeof(int), Dist_gpu+proc_start_b_i*B*pitch_int, pitch, n*sizeof(int), valid_row, hipMemcpyDeviceToHost);
}

__global__ void cal_kernel(int B, int Round, int block_start_x, int block_start_y, int block_width, int block_height, int n, int* Dist_gpu) {
	
	int b_i = block_start_x + blockIdx.x / block_width;
	int b_j = block_start_y + blockIdx.x % block_width;
	
	int inner_round = (B*B-1)/blockDim.x + 1;
	
	//__shared__ int shared_mem = 
	
	for (int k = Round * B; k < (Round +1) * B && k < n; ++k) {

		for(int r=0; r<inner_round; r++){

			int i = b_i * B + (threadIdx.x + r*blockDim.x) / B;
			int j = b_j * B + (threadIdx.x + r*blockDim.x) % B;

			if ((i>=n) | (j>=n)) continue ;
			//if ((Dist_gpu[i*n+k] + Dist_gpu[k*n+j])==73) printf("%d, %d, %d, %d\n", i, j, k, n);
			if (Dist_gpu[i*n+k] + Dist_gpu[k*n+j] < Dist_gpu[i*n+j]) {
				Dist_gpu[i*n+j] = Dist_gpu[i*n+k] + Dist_gpu[k*n+j];
			}
		}
		__syncthreads();
	}
	
}

__global__ void p1_cal_kernel(int B, int Round, int block_start_x, int block_start_y, int block_width, int block_height, int n, int* Dist_gpu, int pitch_int) {
	
	// if(blockIdx.x==0 && threadIdx.x==0 && Round==0){
	// 	for(int i =0; i<n; i++){
	// 		for(int j=0; j<n; j++){
	// 			printf("%d ",Dist_gpu[i*pitch_int+j]);
	// 		}
	// 		printf("\n");
	// 	}
	// }

	int b_i = block_start_x ;
	int b_j = block_start_y ;
	
	//int inner_round = (B*B-1)/blockDim.x + 1;
	
	extern __shared__ int shared_mem[]; 
	int global_i[ROUND_MAX];
	int global_j[ROUND_MAX];
	int inner_i[ROUND_MAX];
	int inner_j[ROUND_MAX];
	
	#pragma unroll
	for(int r=0; r<4; r++){
		inner_i[r] = threadIdx.y + 16 * r;
		inner_j[r] = threadIdx.x;
		//if(inner_i[r]>=B) continue;
		global_i[r] = b_i * B + inner_i[r];
		global_j[r] = b_j * B + inner_j[r];
		if (!((global_i[r]>=n) | (global_j[r]>=n))) 
			shared_mem[inner_i[r]*B + inner_j[r]] = Dist_gpu[global_i[r]*pitch_int + global_j[r]]; 		
	}


	// if(blockIdx.x==0 && threadIdx.x==0){
	// 	for(int i=0; i<B; i++){
	// 		for(int j=0; j<B; j++){
	// 			printf("%d ", shared_mem[i*B+j]);
	// 		}
	// 		printf("\n");
	// 	}
	// }
	// __syncthreads();

	for (int k = 0; k <  B && (k+Round*B) < n; ++k) {
		__syncthreads();

		#pragma unroll
		for(int r=0; r<4; r++){
			//if(inner_i[r]>=B) continue;
			if ((global_i[r]>=n) | (global_j[r]>=n)) continue ;			

			if (shared_mem[inner_i[r]*B+inner_j[r]] > shared_mem[inner_i[r]*B+k] + shared_mem[k*B+inner_j[r]]) {
				shared_mem[inner_i[r]*B+inner_j[r]] = shared_mem[inner_i[r]*B+k] + shared_mem[k*B+inner_j[r]];
			}
		}
		
	}

	#pragma unroll
	for(int r=0; r<4; r++){
		//if(inner_i[r]>=B) continue;
		if (!((global_i[r]>=n) | (global_j[r]>=n))) 
			Dist_gpu[global_i[r]*pitch_int + global_j[r]] = shared_mem[inner_i[r]*B + inner_j[r]];	
	}
	
}


extern __shared__ int shared_mem[]; 
__global__ void p2_cal_kernel(int B, int Round, int n, int* Dist_gpu, int pitch_int) {
	
	int b_i, b_j;
	if(blockIdx.y==0){
		b_i = Round;
		b_j = blockIdx.x + (blockIdx.x>=Round);
	}
	else{
		b_i = blockIdx.x + (blockIdx.x>=Round);
		b_j = Round;
	}
	
	//int inner_round = (B*B-1)/blockDim.x + 1;
	
	
	int global_i[ROUND_MAX];
	int global_j[ROUND_MAX];
	int inner_i[ROUND_MAX];
	int inner_j[ROUND_MAX];
	
	#pragma unroll
	for(int r=0; r<4; r++){
		inner_i[r] = threadIdx.y + 16 * r;
		inner_j[r] = threadIdx.x;
		//if(inner_i[r]>=B) continue;
		global_i[r] = b_i * B + inner_i[r];
		global_j[r] = b_j * B + inner_j[r];
		int global_pivot_i = Round * B + inner_i[r];
		int global_pivot_j = Round * B + inner_j[r];
		if (!((global_i[r]>=n) | (global_j[r]>=n))) 
			shared_mem[inner_i[r]*B + inner_j[r]] = Dist_gpu[global_i[r]*pitch_int + global_j[r]];
		if (!((global_pivot_i>=n) | (global_pivot_j>=n))) 
			shared_mem[inner_i[r]*B + inner_j[r] + B*B] = Dist_gpu[global_pivot_i*pitch_int + global_pivot_j];
	}
	

	for (int k = 0; k <  B && (k+Round*B) < n; ++k) {
		__syncthreads();

		#pragma unroll
		for(int r=0; r<4; r++){
			//if(inner_i[r]>=B) continue;
			if ((global_i[r]>=n) | (global_j[r]>=n)) continue ;

			//if ((Dist_gpu[i*n+k] + Dist_gpu[k*n+j])==73) printf("%d, %d, %d, %d\n", i, j, k, n);
			if (shared_mem[inner_i[r]*B+inner_j[r]] > shared_mem[inner_i[r]*B+k + !blockIdx.y*B*B] + shared_mem[k*B+inner_j[r] + blockIdx.y*B*B]) {
				shared_mem[inner_i[r]*B+inner_j[r]] = shared_mem[inner_i[r]*B+k + !blockIdx.y*B*B] + shared_mem[k*B+inner_j[r] + blockIdx.y*B*B];
			}
			
		}
		
	}
	#pragma unroll
	for(int r=0; r<4; r++){
		//if(inner_i[r]>=B) continue;
		if (!((global_i[r]>=n) | (global_j[r]>=n))) 
			Dist_gpu[global_i[r]*pitch_int + global_j[r]] = shared_mem[inner_i[r]*B + inner_j[r]];
				
	}

	
}

__global__ void p3_cal_kernel(int B, int Round, int k_i, int n, int* Dist_gpu, int pitch_int) {

    /*MPI*/
	int b_i = k_i+blockIdx.x + ((k_i+blockIdx.x)>=Round);
	int b_j = blockIdx.y + (blockIdx.y>=Round);

	__shared__ int shared_mem[8192]; 
	//int inner_round = (B*B-1)/blockDim.x + 1;
		
	int global_i[ROUND_MAX];
	int global_j[ROUND_MAX];
	int inner_i[ROUND_MAX];
	int inner_j[ROUND_MAX];
	int my_dist[ROUND_MAX];
	
	#pragma unroll
	for(int r=0; r<4; r++){
		//if(inner_i[r]>=B) continue;
		inner_i[r] = threadIdx.y + 16 * r;
		inner_j[r] = threadIdx.x;
		global_i[r] = b_i * B + inner_i[r];
		global_j[r] = b_j * B + inner_j[r];
		int row_pivot_i = global_i[r];
		int row_pivot_j = Round * B + inner_j[r];
		int col_pivot_i = Round * B + inner_i[r];
		int col_pivot_j = global_j[r];

		my_dist[r] = Dist_gpu[global_i[r]*pitch_int + global_j[r]];
		shared_mem[inner_i[r]*B + inner_j[r] ] = Dist_gpu[row_pivot_i*pitch_int + row_pivot_j];
		shared_mem[inner_i[r]*B + inner_j[r] + B*B] = Dist_gpu[col_pivot_i*pitch_int + col_pivot_j];
		
	}

	__syncthreads();
	for (int k = 0; k <  B && (k+Round*B) < n; ++k) {
		#pragma unroll
		for(int r=0; r<4; r++){			
			int tmp = shared_mem[inner_i[r]*B+k ] + shared_mem[k*B+inner_j[r] +B*B];
			if (my_dist[r] > tmp) {
				my_dist[r] = tmp;
			}			
		}
	}

	#pragma unroll
	for(int r=0; r<4; r++){
		Dist_gpu[global_i[r]*pitch_int + global_j[r]] = my_dist[r];
		 		
	}

}
